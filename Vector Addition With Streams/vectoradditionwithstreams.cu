#include "hip/hip_runtime.h"
#include	<wb.h>

#define BLOCK_SIZE 256
#define SEG_SIZE 1536


__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
	//@@ Insert code to implement vector addition here	
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx < len)
	{
		out[idx] = in1[idx] + in2[idx];
	}
}

int main(int argc, char ** argv) {
	wbArg_t args;
	int inputLength;

	float * hostInput1;
	float * hostInput2;
	float * hostOutput;

	float * deviceInput0a;
	float * deviceInput1a;
	float * deviceInput2a;
	float * deviceInput3a;

	float * deviceInput0b;
	float * deviceInput1b;
	float * deviceInput2b;
	float * deviceInput3b;

	float * deviceOutput0;
	float * deviceOutput1;
	float * deviceOutput2;
	float * deviceOutput3;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *) malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");
	wbTime_start(GPU, "Allocating GPU memory.");

	hipStream_t stream0, stream1, stream2, stream3;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	//@@ Allocate GPU memory here
	int lengthInBytes = inputLength * sizeof(float);
	int lengthChunk   = SEG_SIZE* sizeof(float);
	
	hipHostRegister(hostInput1,lengthInBytes,0);
	hipHostRegister(hostInput2,lengthInBytes,0);
	hipHostRegister(hostOutput,lengthInBytes,0);

	hipMalloc((void**) &deviceInput0a, lengthChunk);
	hipMalloc((void**) &deviceInput0b, lengthChunk);
	hipMalloc((void**) &deviceInput1a, lengthChunk);
	hipMalloc((void**) &deviceInput1b, lengthChunk);	
	hipMalloc((void**) &deviceInput2a, lengthChunk);
	hipMalloc((void**) &deviceInput2b, lengthChunk);
	hipMalloc((void**) &deviceInput3a, lengthChunk);
	hipMalloc((void**) &deviceInput3b, lengthChunk);

	hipMalloc((void**) &deviceOutput0, lengthChunk);
	hipMalloc((void**) &deviceOutput1, lengthChunk);
	hipMalloc((void**) &deviceOutput2, lengthChunk);
	hipMalloc((void**) &deviceOutput3, lengthChunk);

	wbTime_stop(GPU, "Allocating GPU memory.");
	int n = ceil(((double)inputLength)/SEG_SIZE);

	wbTime_start(GPU, "CUDA Streaming.");
	for(int i=0; i<n*SEG_SIZE; i+=SEG_SIZE*4)
	{
		//@@ Copy memory to the GPU here

		int lengthChunk0 = sizeof(float) * (i+  SEG_SIZE <= inputLength ? SEG_SIZE : inputLength - i);
		int lengthChunk1 = sizeof(float) * (i+2*SEG_SIZE <= inputLength ? SEG_SIZE : inputLength - i - SEG_SIZE);
		int lengthChunk2 = sizeof(float) * (i+3*SEG_SIZE <= inputLength ? SEG_SIZE : inputLength - i - 2*SEG_SIZE);
		int lengthChunk3 = sizeof(float) * (i+4*SEG_SIZE <= inputLength ? SEG_SIZE : inputLength - i - 3*SEG_SIZE);

		if(lengthChunk0>0)
		{
			hipMemcpyAsync(deviceInput0a, hostInput1+i, lengthChunk0, hipMemcpyHostToDevice, stream0);
			hipMemcpyAsync(deviceInput0b, hostInput2+i, lengthChunk0, hipMemcpyHostToDevice, stream0);
		}
		
		if(lengthChunk1>0)
		{
			hipMemcpyAsync(deviceInput1a, hostInput1+i+SEG_SIZE, lengthChunk1, hipMemcpyHostToDevice, stream1);
			hipMemcpyAsync(deviceInput1b, hostInput2+i+SEG_SIZE, lengthChunk1, hipMemcpyHostToDevice, stream1);
		}
		
		if(lengthChunk2>0)
		{
			hipMemcpyAsync(deviceInput2a, hostInput1+i+2*SEG_SIZE, lengthChunk2, hipMemcpyHostToDevice, stream2);
			hipMemcpyAsync(deviceInput2b, hostInput2+i+2*SEG_SIZE, lengthChunk2, hipMemcpyHostToDevice, stream2);
		}

		if(lengthChunk3>0)
		{
			hipMemcpyAsync(deviceInput3a, hostInput1+i+3*SEG_SIZE, lengthChunk3, hipMemcpyHostToDevice, stream3);
			hipMemcpyAsync(deviceInput3b, hostInput2+i+3*SEG_SIZE, lengthChunk3, hipMemcpyHostToDevice, stream3);
		}

		//@@ Launch the GPU Kernel here
		if(lengthChunk0>0)
			vecAdd<<<ceil(((double)lengthChunk0)/(sizeof(float)*BLOCK_SIZE)),BLOCK_SIZE,0,stream0>>>(deviceInput0a, deviceInput0b, deviceOutput0, lengthChunk0/sizeof(float));

		if(lengthChunk1>0)
			vecAdd<<<ceil(((double)lengthChunk1)/(sizeof(float)*BLOCK_SIZE)),BLOCK_SIZE,0,stream1>>>(deviceInput1a, deviceInput1b, deviceOutput1, lengthChunk1/sizeof(float));

		if(lengthChunk2>0)
			vecAdd<<<ceil(((double)lengthChunk2)/(sizeof(float)*BLOCK_SIZE)),BLOCK_SIZE,0,stream2>>>(deviceInput2a, deviceInput2b, deviceOutput2, lengthChunk2/sizeof(float));

		if(lengthChunk3>0)
			vecAdd<<<ceil(((double)lengthChunk3)/(sizeof(float)*BLOCK_SIZE)),BLOCK_SIZE,0,stream3>>>(deviceInput3a, deviceInput3b, deviceOutput3, lengthChunk3/sizeof(float));

		//@@ Copy the GPU memory back to the CPU here
		
		if(lengthChunk0>0)
			hipMemcpyAsync(hostOutput+i, deviceOutput0, lengthChunk0, hipMemcpyDeviceToHost, stream0);

		if(lengthChunk1>0)
			hipMemcpyAsync(hostOutput+i+SEG_SIZE, deviceOutput1, lengthChunk1, hipMemcpyDeviceToHost, stream1);

		if(lengthChunk2>0)
			hipMemcpyAsync(hostOutput+i+2*SEG_SIZE, deviceOutput2, lengthChunk2, hipMemcpyDeviceToHost, stream2);

		if(lengthChunk3>0)
			hipMemcpyAsync(hostOutput+i+3*SEG_SIZE, deviceOutput3, lengthChunk3, hipMemcpyDeviceToHost, stream3);
	}
	hipDeviceSynchronize();
	hipHostUnregister(hostInput1);
	hipHostUnregister(hostInput2);
	hipHostUnregister(hostOutput);
	wbTime_stop(GPU, "CUDA Streaming.");

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here
	hipFree(deviceInput0a);
	hipFree(deviceInput1a);
	hipFree(deviceInput2a);
	hipFree(deviceInput3a);
	
	hipFree(deviceInput0b);
	hipFree(deviceInput1b);
	hipFree(deviceInput2b);
	hipFree(deviceInput3b);

	hipFree(deviceOutput0);
	hipFree(deviceOutput1);
	hipFree(deviceOutput2);
	hipFree(deviceOutput3);

	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}

