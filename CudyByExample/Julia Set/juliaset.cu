#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>


struct hipComplex
{
	float r;
	float i;

	__host__ __device__ hipComplex(float a, float b) : r(a), i(b) { }
	__host__ __device__ float magnitude2() { return r*r + i*i; }
	__host__ __device__ hipComplex operator*(const hipComplex & a) { return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i) ; }
	__host__ __device__ hipComplex operator+(const hipComplex & a) { return hipComplex(r+a.r,i+a.i); } 
};

__host__ __device__ bool julia(int x, int y, int dimX, int dimY)
{
	const float scale = 1.5;
	float jx = scale * (float)(dimX/2.0 - x) / (dimX/2.0);
	float jy = scale * (float)(dimY/2.0 - y) / (dimY/2.0);

	hipComplex c(-0.8,0.156);
	hipComplex a(jx,jy);

	int i = 0;
	for(i=0; i< 200; ++i)
	{
		a = a*a + c;
		if(a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

void cpuJuliaKernel(unsigned char* data, long rows, long cols)
{
	for(int r = 0; r < rows; ++r)
	{
		for(int c = 0; c < cols; ++c)
		{
			data[r*cols+c] = 255* julia(c,r, cols, rows);
		}
	}
}

__global__ void gpuJuliaKernel(unsigned char* data, long rows, long cols)
{
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	int c = blockIdx.x*blockDim.x + threadIdx.x;

	if(r < rows && c < cols)
		data[r*cols+c] = 255 * julia(c,r, cols, rows);
}

int main()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int imgWidth = 1000;
	int imgHeight = 1000;
	cv::Mat1b img(imgHeight,imgWidth);
	cv::Mat1b imgGPU(imgHeight,imgWidth);
	int blockSize = 16;
	
	cpuJuliaKernel(img.data, img.rows, img.cols);

	unsigned char* dImg;
	dim3 grid( ceil(((float)imgWidth)/blockSize), ceil( ((float)imgHeight)/blockSize),1);
	dim3 threads(blockSize, blockSize, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipMalloc((void**) &dImg, imgGPU.rows * imgGPU.cols * sizeof(unsigned char));
	gpuJuliaKernel<<<grid,threads>>>( dImg, imgGPU.rows, imgGPU.cols );
	//hipDeviceSynchronize();
	hipMemcpy(imgGPU.data, dImg, imgGPU.rows*imgGPU.cols*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(dImg);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Time to do Julia: %3.1f ms\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	cv::namedWindow("Julia Set");
	cv::imshow("Julia Set", imgGPU);
	cv::waitKey();
	return 0;
}

