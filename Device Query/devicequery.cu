#include	<wb.h>

//@@ The purpose of this code is to become familiar with the submission 
//@@ process. Do not worry if you do not understand all the details of 
//@@ the code.

int main(int argc, char ** argv) {
    int deviceCount;

    wbArg_read(argc, argv);

    hipGetDeviceCount(&deviceCount);

    wbTime_start(GPU, "Getting GPU Data."); //@@ start a timer

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                wbLog(TRACE, "No CUDA GPU has been detected");
                return -1;
            } else if (deviceCount == 1) {
                //@@ WbLog is a provided logging API (similar to Log4J).
                //@@ The logging function wbLog takes a level which is either
                //@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
                //@@ message to be printed.
                wbLog(TRACE, "There is 1 device supporting CUDA");
            } else {
                wbLog(TRACE, "There are ", deviceCount, " devices supporting CUDA");
            }
        }

        wbLog(TRACE, "Device ", dev, " name: ", deviceProp.name);
        wbLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".", deviceProp.minor);
        wbLog(TRACE, " Maximum global memory size: ", deviceProp.totalGlobalMem);
        wbLog(TRACE, " Maximum constant memory size: ", deviceProp.totalConstMem);
        wbLog(TRACE, " Maximum shared memory size per block: ", deviceProp.sharedMemPerBlock);
        wbLog(TRACE, " Maximum block dimensions: ", deviceProp.maxThreadsDim[0], " x ",
                                                    deviceProp.maxThreadsDim[1], " x ",
                                                    deviceProp.maxThreadsDim[2]);
        wbLog(TRACE, " Maximum grid dimensions: ", deviceProp.maxGridSize[0], " x ",
                                                   deviceProp.maxGridSize[1], " x ",
                                                   deviceProp.maxGridSize[2]);
        wbLog(TRACE, " Warp size: ", deviceProp.warpSize);

		
		printf("Computational Capabilities: %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("Maximum global memory size: %d\n", deviceProp.totalGlobalMem);
		printf("Maximum constant memory size: %d\n", deviceProp.totalConstMem);
		printf("Maximum shared memory size per block: %d\n", deviceProp.sharedMemPerBlock);
		printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
		printf("Maximum block dimensions: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("Maximum grid dimensions: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("Warp size: %d\n", deviceProp.warpSize);

    }

    wbTime_stop(GPU, "Getting GPU Data."); //@@ stop the timer

    return 0;
}

